// compute.cu
//
// driver and kernel call


#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 256
 
__global__ void compute_d (int *a_d, int *b_d, int *c_d, int n)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < n) {
		a_d[x] = x % 10 + 1;

        if (x < (n / 2))
        {
            b_d[x] = x + 1;
            //j = x + 1;
        } else if (x == (n / 2) && 0 < n % 2)
        {
            b_d[x] = x + 1;
        } else
        {
            b_d[x] = (n - x);
        }
		c_d[x] = a_d[x] * b_d[x];
	}
		
}

extern "C" void computeArr (int *c, int arraySize)
{
	int *a_d, *b_d, *c_d;

	hipMalloc ((void**) &a_d, sizeof(int) * arraySize);
	hipMalloc ((void**) &b_d, sizeof(int) * arraySize);
	hipMalloc ((void**) &c_d, sizeof(int) * arraySize);

	compute_d <<< ceil((float) arraySize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (a_d, b_d, c_d, arraySize);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	hipMemcpy (c, c_d, sizeof(int) * arraySize, hipMemcpyDeviceToHost);
	hipFree (a_d);
	hipFree (b_d);
	hipFree (c_d);
}

